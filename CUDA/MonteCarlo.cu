#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <random>
#include <chrono>

// CUDA kernel for Monte Carlo simulation
__global__ void monte_carlo_pi(const float* random_x, const float* random_y, int* count, int num_points) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < num_points) {
        float x = random_x[id];
        float y = random_y[id];
        if (x * x + y * y <= 1.0f) {
            atomicAdd(count, 1);
        }
    }
}

int main() {
    const int num_points = 100000000;
    std::vector<float> random_x(num_points);
    std::vector<float> random_y(num_points);
    int count = 0;

    // Generate random points
    std::mt19937 rng(std::random_device{}());
    std::uniform_real_distribution<float> dist(0.0f, 1.0f);
    for (int i = 0; i < num_points; ++i) {
        random_x[i] = dist(rng);
        random_y[i] = dist(rng);
    }

    // Allocate device memory
    float *d_random_x, *d_random_y;
    int *d_count;
    hipMalloc(&d_random_x, sizeof(float) * num_points);
    hipMalloc(&d_random_y, sizeof(float) * num_points);
    hipMalloc(&d_count, sizeof(int));

    // Copy data to device
    hipMemcpy(d_random_x, random_x.data(), sizeof(float) * num_points, hipMemcpyHostToDevice);
    hipMemcpy(d_random_y, random_y.data(), sizeof(float) * num_points, hipMemcpyHostToDevice);
    hipMemcpy(d_count, &count, sizeof(int), hipMemcpyHostToDevice);

    std::cout << "Running CUDA Monte Carlo simulation to estimate Pi..." << std::endl;

    // Measure execution time
    auto startTime = std::chrono::high_resolution_clock::now();

    // Launch kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (num_points + threadsPerBlock - 1) / threadsPerBlock;
    monte_carlo_pi<<<blocksPerGrid, threadsPerBlock>>>(d_random_x, d_random_y, d_count, num_points);

    // Copy result back to host
    hipMemcpy(&count, d_count, sizeof(int), hipMemcpyDeviceToHost);

    auto endTime = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(endTime - startTime);

    std::cout << "CUDA Monte Carlo completed in " << duration.count() << " ms" << std::endl;

    // Calculate Pi
    float pi = 4.0f * count / num_points;
    std::cout << "Estimated value of Pi: " << pi << std::endl;

    // Free device memory
    hipFree(d_random_x);
    hipFree(d_random_y);
    hipFree(d_count);

    return 0;
}