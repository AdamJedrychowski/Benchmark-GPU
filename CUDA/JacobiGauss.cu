#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <cmath>
#include <cstdlib>
#include <chrono>
#include "../Matrix.h"

// CUDA kernel for Jacobi Iteration
__global__ void jacobiKernel(const float* A, const float* b, float* x, float* x_new, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        float sum = 0.0f;
        for (int j = 0; j < n; ++j) {
            if (i != j) {
                sum += A[i * n + j] * x[j];
            }
        }
        x_new[i] = (b[i] - sum) / A[i * n + i];
    }
}

int main() {
    srand(static_cast<unsigned int>(time(0)));
    int n = 150;
    Matrix<float> A = Matrix<float>::generateMatrixSystemEquations(n);
    std::vector<float> b(n);
    for (int i = 0; i < n; ++i) {
        b[i] = static_cast<float>(rand() % 1000000 + 1);
    }
    std::vector<float> x(n, 0.0f), x_new(n, 0.0f);

    int maxIterations = 1000;
    float tolerance = 1e-6;

    float *d_A, *d_b, *d_x, *d_x_new;

    // Allocate device memory
    hipMalloc(&d_A, sizeof(float) * n * n);
    hipMalloc(&d_b, sizeof(float) * n);
    hipMalloc(&d_x, sizeof(float) * n);
    hipMalloc(&d_x_new, sizeof(float) * n);

    // Copy data to device
    hipMemcpy(d_A, A.data(), sizeof(float) * n * n, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b.data(), sizeof(float) * n, hipMemcpyHostToDevice);
    hipMemcpy(d_x, x.data(), sizeof(float) * n, hipMemcpyHostToDevice);

    std::cout << "Starting Jacobi Iteration..." << std::endl;
    auto startTime = std::chrono::high_resolution_clock::now();

    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

    for (int iter = 0; iter < maxIterations; ++iter) {
        jacobiKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_b, d_x, d_x_new, n);
        hipDeviceSynchronize();

        hipMemcpy(x_new.data(), d_x_new, sizeof(float) * n, hipMemcpyDeviceToHost);

        float error = 0.0f;
        for (int i = 0; i < n; ++i) {
            error += std::abs(x_new[i] - x[i]);
        }
        if (error < tolerance) {
            break;
        }

        hipMemcpy(d_x, d_x_new, sizeof(float) * n, hipMemcpyDeviceToDevice);
    }

    auto endTime = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> duration = endTime - startTime;
    std::cout << "Jacobi Iteration completed in " << duration.count() << " seconds." << std::endl;
    std::cout << "Solution: ";
    for (int i = 0; i < n; ++i) {
        std::cout << x_new[i] << " ";
    }

    // Free device memory
    hipFree(d_A);
    hipFree(d_b);
    hipFree(d_x);
    hipFree(d_x_new);

    return 0;
}