#define _USE_MATH_DEFINES
#include <iostream>
#include <vector>
#include <cmath>
#ifndef M_PI
#define M_PI 3.14159265358979323846
#endif
#include <chrono>
#include <hip/hip_runtime.h>

// Parametry fizyczne i numeryczne
const double alpha = 0.01;   // współczynnik przewodzenia ciepła
const double L = 1.0;        // długość pręta
const double T = 1.0;        // czas całkowity

const int Nx = 20;           // liczba punktów przestrzennych
const int Nt = 1000;         // liczba kroków czasowych

const double dx = L / (Nx - 1);
const double dt = T / Nt;

const double r = alpha * dt / (dx * dx);  // parametr stabilności

__global__ void heatEquationKernel(double* u, double* u_next, double r, int Nx) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i > 0 && i < Nx - 1) {
        u_next[i] = u[i] + r * (u[i + 1] - 2 * u[i] + u[i - 1]);
    }
}

int main() {
    std::vector<double> u(Nx);
    std::vector<double> u_next(Nx);

    // Inicjalizacja warunku początkowego
    for (int i = 0; i < Nx; ++i) {
        double x = i * dx;
        u[i] = sin(M_PI * x);
    }

    double *d_u, *d_u_next;
    hipMalloc(&d_u, Nx * sizeof(double));
    hipMalloc(&d_u_next, Nx * sizeof(double));

    hipMemcpy(d_u, u.data(), Nx * sizeof(double), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (Nx + blockSize - 1) / blockSize;

    std::cout << "Running CUDA heat equation solver..." << std::endl;
    auto startTime = std::chrono::high_resolution_clock::now();

    for (int n = 0; n < Nt; ++n) {
        heatEquationKernel<<<numBlocks, blockSize>>>(d_u, d_u_next, r, Nx);
        hipMemcpy(d_u, d_u_next, Nx * sizeof(double), hipMemcpyDeviceToDevice);
    }

    hipMemcpy(u.data(), d_u, Nx * sizeof(double), hipMemcpyDeviceToHost);

    auto endTime = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(endTime - startTime);
    std::cout << "CUDA heat equation solver completed in " << duration.count() << " ms" << std::endl;

    std::cout << "x,u_final\n";
    for (int i = 0; i < Nx; ++i) {
        double x = i * dx;
        std::cout << x << "," << u[i] << "\n";
    }

    hipFree(d_u);
    hipFree(d_u_next);

    return 0;
}